#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

#include <glad/glad.h>
#include <GLFW/glfw3.h>
#include <time.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "particle_renderer.hpp"
#include "nbody.cuh"
#include <hiprand/hiprand_kernel.h>

#define DIM 512
#define G 6.674e-11


void CHECK_CUDA(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    }
}

void setDevice() {
    hipDeviceProp_t prop;
    int dev;

    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 1;
    prop.minor = 0;
    hipChooseDevice(&dev, &prop);

    cudaGLSetGLDevice(dev);

    std::cout << "Using device " << dev << std::endl;
}

extern "C" void launchInitKernel(unsigned int numBlocks, unsigned int threadsPerBlock, float3* positions, char * type) {
    hipError_t cudaStatus;

    unsigned int sqrtThreads = (unsigned int)sqrt(threadsPerBlock);
    static hiprandState *states = NULL;
    hipGetErrorString(hipMalloc((void **)&states, sizeof(hiprandState) *
                sqrtThreads * sqrtThreads*numBlocks));
    // unsigned int sqrtThreads = threadsPerBlock;
    dim3 threads(sqrtThreads, sqrtThreads);
    std::cout<< type<< std::endl;
    float * bounds, *bounds_h;
    hipMalloc(&bounds, sizeof(float) *6);
    bounds_h = (float *) malloc(sizeof(float)* 6);
    if (strcmp(type, "Lorenz")==0){
        bounds_h[0] = -25.; bounds_h[1] = 20.; bounds_h[2] = 5.; bounds_h[3] = 40.; bounds_h[4] = 10.; bounds_h[5] = 40.;
        hipMemcpy(bounds, bounds_h, sizeof(float)*6, hipMemcpyHostToDevice);
    }
    if (strcmp(type, "Pickover")==0){
        bounds_h[0] = -25.; bounds_h[1] = 20.; bounds_h[2] = 5.; bounds_h[3] = 40.; bounds_h[4] = 10.; bounds_h[5] = 40.;
        hipMemcpy(bounds, bounds_h, sizeof(float)*6, hipMemcpyHostToDevice);

    }
    if (strcmp(type, "Chen")==0){
        bounds_h[0] = -15.; bounds_h[1] = 15.; bounds_h[2] = -15.; bounds_h[3] = 15.; bounds_h[4] = 10.; bounds_h[5] = 40.;
        hipMemcpy(bounds, bounds_h, sizeof(float)*6, hipMemcpyHostToDevice);
    }
    if (strcmp(type, "Izawa")==0){
        bounds_h[0] = -2.; bounds_h[1] =2.; bounds_h[2] = -2.; bounds_h[3] = 2.; bounds_h[4] = -2.; bounds_h[5] = 0.;
        hipMemcpy(bounds, bounds_h, sizeof(float)*6, hipMemcpyHostToDevice);
    }
    std::cout<< bounds_h[5]<< std::endl;
    plane<<<numBlocks, threads>>>(states, positions, bounds);
    cudaStatus = hipGetLastError();


    if (cudaStatus != hipSuccess) {
        std::cerr << "Error launching initialization kernel: " << hipGetErrorString(cudaStatus) << std::endl;
    }
}

void launchGravityKernel(unsigned int numBlocks, unsigned int threadsPerBlock, float3* positions, float3* velocities) {
    hipError_t cudaStatus;
    unsigned int sqrtThreads = (unsigned int)sqrt(threadsPerBlock);
    // unsigned int sqrtThreads = threadsPerBlock;
    dim3 threads(sqrtThreads, sqrtThreads);

    float mass = 100000.0; //kg
    // float dt = 100.0; //seconds
    float dt = 0.005;//0.001; //seconds
    if (strcmp(type, "Lorenz")==0){
    lorenzKernel<<<numBlocks, threads>>>(positions, velocities, mass, dt);
    }
    else if (strcmp(type, "Pickover")==0){
  //      pickoverKernel<<<numBlocks, threads>>>(positions, velocities, mass, dt);
    }
    else if(strcmp(type, "Chen")==0){
        chenKernel<<<numBlocks, threads>>>(positions, velocities, mass, dt);
    }
    else if(strcmp(type, "Izawa")==0){
        IsawaKernel<<<numBlocks, threads>>>(positions, velocities, mass, dt);
    }
    cudaStatus = hipGetLastError();

    if (cudaStatus != hipSuccess) {
        std::cerr << "Error launching kernel: " << hipGetErrorString(cudaStatus) << std::endl;
    }
}

extern "C" __global__ void plane(hiprandState *states, float3* positions, float * bounds) {
    // unsigned int id = threadIdx.x + threadIdx.y * blockDim.x;
    int tid = threadIdx.x;
    int col_offset = blockDim.x * blockDim.y * blockIdx.x;
    int row_offset = gridDim.x * blockIdx.y * blockDim.x * blockDim.y + blockDim.x * threadIdx.y;
    int id = tid + col_offset + row_offset;
    hiprandState *state = states + tid;
    hiprand_init(9384, id, 0, state);

    // hiprandState *d_state;
    // hipMalloc(&d_state, sizeof(hiprandState));
    // unsigned *x_result, *h_result;
    // unsigned *d_max_rand_int, *h_max_rand_int, *d_min_rand_int, *h_min_rand_int;
    // hipMalloc(&d_result, (MAX-MIN+1) * sizeof(unsigned));
    // h_result = (unsigned *)malloc((MAX-MIN+1)*sizeof(unsigned));
    // hipMalloc(&d_max_rand_int, sizeof(unsigned));
    // h_max_rand_int = (unsigned *)malloc(sizeof(unsigned));
    // hipMalloc(&d_min_rand_int, sizeof(unsigned));
    // h_min_rand_int = (unsigned *)malloc(sizeof(unsigned));
    // hipMemset(d_result, 0, (MAX-MIN+1)*sizeof(unsigned));
    // setup_kernel<<<1,1>>>(d_state);

    // *h_max_rand_int = MAX;
    // *h_min_rand_int = MIN;
    // hipMemcpy(d_max_rand_int, h_max_rand_int, sizeof(unsigned), hipMemcpyHostToDevice);
    // hipMemcpy(d_min_rand_int, h_min_rand_int, sizeof(unsigned), hipMemcpyHostToDevice);
    // generate_kernel<<<1,1>>>(d_state, ITER, d_max_rand_int, d_min_rand_int, d_result);
    // hipMemcpy(h_result, d_result, (MAX-MIN+1) * sizeof(unsigned), hipMemcpyDeviceToHost);
    // printf("Bin:    Count: \n");
    // for (int i = MIN; i <= MAX; i++)
    //     printf("%d    %d\n", i, h_result[i-MIN]);
 
    // positions[id].x = (threadIdx.x) * 50.0;
    // positions[id].y = (blockIdx.x * 50.0) - 1000.0 ;
    // positions[id].z = (threadIdx.y) * 50.0;
   
        float rand = hiprand_uniform(state);
        positions[id].x = rand*(bounds[1]-bounds[0])+bounds[0];
        rand = hiprand_uniform(state);
        positions[id].y = rand*(bounds[3]-bounds[2])+bounds[2];
        rand = hiprand_uniform(state);
        positions[id].z = rand*(bounds[5]-bounds[4])+bounds[4];



    // positions[id].x = 10 * threadIdx.x / 32 - 5;
    // positions[id].y = 10.* threadIdx.y / 32 - 5;
    // positions[id].z = 10. * blockIdx.x / 4 - 5;
}

__global__ void gravityKernel(float3* positions, float3* d_velocity, float mass, float dt) {
    int tid = threadIdx.x;
    int col_offset = blockDim.x * blockDim.y * blockIdx.x;
    int row_offset = gridDim.x * blockIdx.y * blockDim.x * blockDim.y + blockDim.x * threadIdx.y;
    int i = tid + col_offset + row_offset;
    const float3 d0_i = positions[i];
    float3 a = {0, 0, 0};

    for (int j = 0; j < blockDim.x * blockDim.y * gridDim.x; j++) {
        if (j == i) continue;

        const float3 d0_j = positions[j];
        float3 r_ij;
        r_ij.x = d0_i.x - d0_j.x;
        r_ij.y = d0_i.y - d0_j.y;
        r_ij.z = d0_i.z - d0_j.z;

        float r_squared = (r_ij.x * r_ij.x) + (r_ij.y * r_ij.y) + (r_ij.z * r_ij.z);

        float F_coef = -G * mass / r_squared;

        a.x += F_coef * r_ij.x * rsqrt(r_squared);
        a.y += F_coef * r_ij.y * rsqrt(r_squared);
        a.z += F_coef * r_ij.z * rsqrt(r_squared);

       
    }   
        const float3 v0_i = d_velocity[i];
        d_velocity[i].x = v0_i.x + (a.x * dt);
        d_velocity[i].y = v0_i.y + (a.y * dt);
        d_velocity[i].z = v0_i.z + (a.z * dt);

        // positions[i].x = (d0_i.x + v0_i.x * dt + a.x * dt * dt / 2.0);
        // positions[i].y = (d0_i.y + v0_i.y * dt + a.y * dt * dt / 2.0);
        // positions[i].z = (d0_i.z + v0_i.z * dt + a.z * dt * dt / 2.0);
}

extern "C" __global__ void lorenzKernel(float3* positions, float3* d_velocity, float mass, float dt) {
    int tid = threadIdx.x;
    int col_offset = blockDim.x * blockDim.y * blockIdx.x;
    int row_offset = gridDim.x * blockIdx.y * blockDim.x * blockDim.y + blockDim.x * threadIdx.y;
    int i = tid + col_offset + row_offset;
    const float3 d0_i = positions[i];
    float3 a = {0, 0, 0};

    const float sigma = 10.;
    const float rho = 28.;
    const float beta = 8./3.;

    const float3 v0_i = d_velocity[i];
    
    d_velocity[i].x = sigma * (d0_i.y - d0_i.x);
    d_velocity[i].y = d0_i.x * (rho - d0_i.z) - d0_i.y;
    d_velocity[i].z = d0_i.x * d0_i.y - beta * d0_i.z;

    positions[i].x = (d0_i.x + d_velocity[i].x * dt);
    positions[i].y = (d0_i.y + d_velocity[i].y * dt);
    positions[i].z = (d0_i.z + d_velocity[i].z * dt);
}


// __global__ void pickoverKernel(float3* positions, float3* d_velocity, float mass, float dt) {
//     int tid = threadIdx.x;
//     int col_offset = blockDim.x * blockDim.y * blockIdx.x;
//     int row_offset = gridDim.x * blockIdx.y * blockDim.x * blockDim.y + blockDim.x * threadIdx.y;
//     int i = tid + col_offset + row_offset;

//     // parameters 1, 1.8, 0.71, 1.51
//     auto a = 1.;
//     auto b = 1.8; 
//     auto c = 0.71;
//     auto d = 1.51;

//     const float3 d0_i = positions[i];
//     const float3 v0_i = d_velocity[i];



//     float new_x = sin(a * d0_i.y) - d0_i.z * cos(b * d0_i.x);
//     float new_y = d0_i.z * sin(c * d0_i.x) - cos(d * d0_i.y);
//     float new_z = sin(d0_i.x);
//     d_velocity[i].x = (new_x-d0_i.x)*300;
//     d_velocity[i].y = (new_y-d0_i.y)*300;
//     d_velocity[i].z = (new_z-d0_i.z)*300;

//     positions[i].x = new_x;
//     positions[i].y = new_y;
//     positions[i].z = new_z;
//}


__global__ void chenKernel(float3* positions, float3* d_velocity, float mass, float dt) {
    int tid = threadIdx.x;
    int col_offset = blockDim.x * blockDim.y * blockIdx.x;
    int row_offset = gridDim.x * blockIdx.y * blockDim.x * blockDim.y + blockDim.x * threadIdx.y;
    int i = tid + col_offset + row_offset;
    const float3 d0_i = positions[i];

    const float sigma = -0.38;
    const float alpha = 5.;
    const float beta = -10.;

    const float3 v0_i = d_velocity[i];
    d_velocity[i].x = alpha *  d0_i.x- d0_i.y*d0_i.z;
    d_velocity[i].y = beta*d0_i.y+d0_i.x*d0_i.z;
    d_velocity[i].z = sigma*d0_i.z+d0_i.x*d0_i.y/3.;

    positions[i].x = (d0_i.x + d_velocity[i].x * dt);
    positions[i].y = (d0_i.y + d_velocity[i].y * dt);
    positions[i].z = (d0_i.z + d_velocity[i].z * dt);
}

__global__ void IsawaKernel(float3* positions, float3* d_velocity, float mass, float dt) {
    int tid = threadIdx.x;
    int col_offset = blockDim.x * blockDim.y * blockIdx.x;
    int row_offset = gridDim.x * blockIdx.y * blockDim.x * blockDim.y + blockDim.x * threadIdx.y;
    int i = tid + col_offset + row_offset;
    const float3 d0_i = positions[i];

    const float a = 0.95;
    const float b = 0.7;
    const float c = 0.6;
    const float d = 3.5;
    const float e = 0.25;
    const float f = 0.1;

    const float3 v0_i = d_velocity[i];
    float new_x = (d0_i.z-b)*d0_i.x-d*d0_i.y;
    float new_y = d*d0_i.x + (d0_i.z-b)*d0_i.y;
    float new_z = c+a*d0_i.z-d0_i.z*d0_i.z*d0_i.z/3.-(d0_i.x*d0_i.x+d0_i.y*d0_i.y)*(1+e*d0_i.z)+f*d0_i.z*d0_i.x*d0_i.x*d0_i.x;
    
    d_velocity[i].x = new_x;
    d_velocity[i].y = new_y;
    d_velocity[i].z = new_z;

    positions[i].x = (d0_i.x + d_velocity[i].x * dt);
    positions[i].y = (d0_i.y + d_velocity[i].y * dt);
    positions[i].z = (d0_i.z + d_velocity[i].z * dt);

    d_velocity[i].x = new_x*100;
    d_velocity[i].y = new_y*100;
    d_velocity[i].z = new_z*100;
}
